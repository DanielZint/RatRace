#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cfloat>
#include <cstdint>

#include "hip/hip_math_constants.h"

// Open Mesh
#include "MeshHeader.h"
// Size Function
#include "../BackgroundGrid/SizeGrid.h"

/* Keep NQ = 8 for two dimensional meshes! This value was chosen because it gives optimal
performance considering a warp-size of 32 because NQ = 8 results in 8 * 8 = 64 nodes
which is double the warp size. Each vertex is computed using one warp where each warp
computes two grid nodes.
Another implementation used 2 warps for one grid but it was slower as syncthreads is
too expensive.
*/
// Size of Quality Mesh
constexpr int NQ = 8;
// number of refinement steps within DMO
constexpr int  DMO_DEPTH = 3;
// double the maximal number of allowed vertices on the one-ring neighborhood
constexpr int  MAX_ONE_RING_SIZE = 64;


// Error output
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		//fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		fprintf(stderr, "GPUassert: %s. Line %d\n", hipGetErrorString(code), line);
		if (abort) exit(code);
	}
}


typedef union {
	float floats[2];                 // floats[0] = lowest
	int32_t ints[2];                     // ints[1] = lowIdx
	uint64_t ulong;    // for atomic update
} my_atomics;

__device__ uint64_t my_atomicArgMax(uint64_t* address, float val, int32_t idx)
{
	my_atomics loc, newValue;
	loc.floats[0] = val;
	loc.ints[1] = idx;
	newValue.ulong = *address;
	while (newValue.floats[0] < val)
		newValue.ulong = atomicCAS(address, newValue.ulong, loc.ulong);
		
	return newValue.ulong;
}

struct Vertex {
	int oneRingID;
	int n_oneRing;
	int id;				// own vertex id
};

__device__ __forceinline__ float conditionQuad( const real_cu p[4][2] ) {
	real_cu e[4][2];
	real_cu e_length_squared[4];

	for( int i = 0; i < 4; ++i ) {
		int j = ( i + 1 ) % 4;
		e[i][0] = p[j][0] - p[i][0];
		e[i][1] = p[j][1] - p[i][1];

		e_length_squared[i] = e[i][0] * e[i][0] + e[i][1] * e[i][1];
	}
	
	float det0 = e[0][1] * e[3][0] - e[0][0] * e[3][1];
	float det1 = e[1][1] * e[0][0] - e[1][0] * e[0][1];
	float det3 = e[3][1] * e[2][0] - e[3][0] * e[2][1];

	float det = fminf( det0, det1 );
	det = fminf( det, det3 );

	if( det < 0 )
		return det;

	float c0 = 2 * det0 / ( e_length_squared[0] + e_length_squared[3] );
	float c1 = 2 * det1 / ( e_length_squared[1] + e_length_squared[0] );
	float c3 = 2 * det3 / ( e_length_squared[3] + e_length_squared[2] );

	float c = fminf( c0, c1 );
	c = fminf( c, c3 );

	return c;
}

__device__ __forceinline__ float computeConditionQuality( const int n_oneRing, const real_cu oneRing[MAX_ONE_RING_SIZE], const real_cu p[2] ) {
	float q = FLT_MAX;
	for( int k = 0; k < n_oneRing - 1; k += 2 ) {
		real_cu v[4][2] = { { p[0], p[1] },{ oneRing[2 * k], oneRing[2 * k + 1] },{ oneRing[2 * ( k + 1 )], oneRing[2 * ( k + 1 ) + 1] },{ oneRing[2 * ( k + 2 )], oneRing[2 * ( k + 2 ) + 1] } };
		q = fminf( q, conditionQuad( v ) );
	}
	return q;
}

__device__ __forceinline__ float computeLaplaceConditionQuality( const int n_oneRing, const real_cu oneRing[MAX_ONE_RING_SIZE], const real_cu p[2] ) {

	float q = FLT_MAX;

	// compute laplace point
	real_cu lp[2] = { 0,0 };
	for( int k = 0; k < n_oneRing - 1; ++k ) {
		lp[0] += oneRing[2 * k];
		lp[1] += oneRing[2 * k + 1];
	}
	lp[0] /= ( n_oneRing - 1 );
	lp[1] /= ( n_oneRing - 1 );
	lp[0] = p[0] - lp[0];
	lp[1] = p[1] - lp[1];

	for( int k = 0; k < n_oneRing - 1; k += 2 ) {
		real_cu v[4][2] = { { p[0], p[1] },{ oneRing[2 * k], oneRing[2 * k + 1] },{ oneRing[2 * ( k + 1 )], oneRing[2 * ( k + 1 ) + 1] },{ oneRing[2 * ( k + 2 )], oneRing[2 * ( k + 2 ) + 1] } };
		q = fminf( q, conditionQuad( v ) );
	}
	
	if( q < 0.5 )
		return q;
	else
		return 0.5f + 1.f / ( lp[0] * lp[0] + lp[1] * lp[1] + 1 );
}

template<int type = 0>
__device__ __forceinline__ float quality(const int n_oneRing, const real_cu oneRing[MAX_ONE_RING_SIZE], const real_cu p[2], const int q_crit) {
	if constexpr( type == 0 ) {
		return computeConditionQuality( n_oneRing, oneRing, p );
	} 
	if constexpr( type == 1 ) {
		return computeLaplaceConditionQuality( n_oneRing, oneRing, p );
	}
	return -1;
}

template<int type = 0>
__global__ void optimizeHierarchical(int* coloredVertexIDs, const int cOff, const Vertex* vertices, real_cu* vertexPos, int* oneRingVec, const real_cu affineFactor, const real_cu grid_scale) {
	const int i1 = threadIdx.x / NQ;
	const int j1 = threadIdx.x % NQ;

	const int i2 = (threadIdx.x + NQ * NQ / 2) / NQ;
	const int j2 = (threadIdx.x + NQ * NQ / 2) % NQ;

	const Vertex& v = vertices[coloredVertexIDs[cOff + blockIdx.x]];

	float q = -FLT_MAX;

	__shared__ real_cu xPos, yPos;
	__shared__ real_cu maxDistx;
	__shared__ real_cu maxDisty;

	__shared__ my_atomics argMaxVal;
	argMaxVal.floats[0] = -FLT_MAX;
	argMaxVal.ints[1] = NQ*NQ;

	__shared__ real_cu oneRing[MAX_ONE_RING_SIZE];

	// min/max search + loading oneRing
	if (threadIdx.x == 0) {
		maxDistx = -FLT_MAX;
		maxDisty = -FLT_MAX;

		for (int k = 0; k < v.n_oneRing - 1; ++k) {
			real_cu oneRingX = vertexPos[2 * oneRingVec[v.oneRingID + k]];
			real_cu oneRingY = vertexPos[2 * oneRingVec[v.oneRingID + k] + 1];
			oneRing[2 * k] = oneRingX;
			oneRing[2 * k + 1] = oneRingY;

			real_cu xDist = abs(vertexPos[2 * v.id] - oneRingX);
			real_cu yDist = abs(vertexPos[2 * v.id + 1] - oneRingY);

			maxDistx = fmaxf(maxDistx, xDist);
			maxDisty = fmaxf(maxDisty, yDist);
		}
		
		oneRing[2 * v.n_oneRing - 2] = vertexPos[2 * oneRingVec[v.oneRingID + v.n_oneRing - 1]];
		oneRing[2 * v.n_oneRing - 1] = vertexPos[2 * oneRingVec[v.oneRingID + v.n_oneRing - 1] + 1];

		xPos = vertexPos[2 * v.id];
		yPos = vertexPos[2 * v.id + 1];
	}
	__syncwarp();

	// start depth iteration
	real_cu depth_scale = grid_scale;
	real_cu argMax = 0;
	for (int depth = 0; depth < DMO_DEPTH; ++depth) {

		real_cu xMax, xMin, yMax, yMin;
		xMax = xPos + depth_scale * maxDistx;
		xMin = xPos - depth_scale * maxDistx;
		yMax = yPos + depth_scale * maxDisty;
		yMin = yPos - depth_scale * maxDisty;


		real_cu pos_i1 = affineFactor * (i1 * xMin + (NQ - 1 - i1) * xMax);
		real_cu pos_j1 = affineFactor * (j1 * yMin + (NQ - 1 - j1) * yMax);
		real_cu pos_i2 = affineFactor * (i2 * xMin + (NQ - 1 - i2) * xMax);
		real_cu pos_j2 = affineFactor * (j2 * yMin + (NQ - 1 - j2) * yMax);

		real_cu p1[2] = { pos_i1, pos_j1 };
		float q1 = quality<type>( v.n_oneRing, oneRing, p1, 8 );
		real_cu p2[2] = { pos_i2, pos_j2 };
		float q2 = quality<type>( v.n_oneRing, oneRing, p2, 8 );

		if (q1 > q2) {
			q = q1;
			argMax = 1;
		}
		else {
			q = q2;
			argMax = 2;
		}
		__syncwarp();
		my_atomicArgMax( (uint64_t *)&( argMaxVal.ulong ), q, i1 * NQ + j1 );

		real_cu pCurrent[2] = { xPos, yPos };
		float qOld = quality(v.n_oneRing, oneRing, pCurrent, 8);
		if (i1 * NQ + j1 == argMaxVal.ints[1] && qOld < q) {
			if (argMax == 1) {
				xPos = pos_i1;
				yPos = pos_j1;
			}
			else {
				xPos = pos_i2;
				yPos = pos_j2;
			}
		}
		
		//depth dependent scaling factor
		depth_scale = depth_scale * (2.f / (NQ - 1));
	}

	// set new position if it is better than the old one
	real_cu pOld[2] = { vertexPos[2 * v.id] , vertexPos[2 * v.id + 1] };
	float qOld = quality( v.n_oneRing, oneRing, pOld, 8 );
	if (i1 * NQ + j1 == argMaxVal.ints[1] && qOld < q) {
		vertexPos[2 * v.id] = xPos;
		vertexPos[2 * v.id + 1] = yPos;
	}
}

struct UniformGrid
{
	int nx, ny;
	real_cu hx, hy, xMin, yMin, xMax, yMax;
};

inline void copyOpenMeshData( PolyMesh& mesh, real_cu* vertexPos, Vertex* vertices, int* oneRingVec) {

	int interior_counter = 0;
	int oneRing_counter = 0;
	for( auto vh : mesh.vertices() ) {
		auto p = mesh.point( vh );

		vertexPos[2 * vh.idx()] = p[0];
		vertexPos[2 * vh.idx() + 1] = p[1];

		if( !vh.is_boundary() ) {
			// fill vertex struct

			Vertex& v = vertices[interior_counter];
			v.id = vh.idx();

			v.n_oneRing = vh.valence() * 2 + 1;

			v.oneRingID = oneRing_counter;

			LOG_ASSERT( v.n_oneRing <= MAX_ONE_RING_SIZE / 2 );

			auto heh = vh.out();
			auto heh_init = heh;

			do {
				oneRingVec[oneRing_counter++] = heh.to().idx();
				heh = heh.next();
				oneRingVec[oneRing_counter++] = heh.to().idx();
				heh = heh.next().next().opp();
			} while( heh != heh_init );

			oneRingVec[oneRing_counter] = heh.to().idx();
			++oneRing_counter;
			++interior_counter;
		}
	}
}

inline void createColoring( PolyMesh& mesh, const int n_free_vertices, int** coloredVertexIDs, std::vector<int>& colorOffset) {

	// create coloring scheme
	std::vector<int>colorScheme(mesh.n_vertices(), -1);
	int colorSchemeIt = 0;

	// set boundarys to a value that can be ignored
	for (auto v_it = mesh.vertices_begin(); v_it != mesh.vertices_end(); ++v_it) {
		if (mesh.is_boundary(*v_it)) {
			colorScheme[v_it->idx()] = -2;
		}
	}

	while (std::find(colorScheme.begin(), colorScheme.end(), -1) != colorScheme.end()) {
		for (auto v_it = mesh.vertices_begin(); v_it != mesh.vertices_end(); ++v_it) {
	
			if (colorScheme[v_it->idx()] != -1) { continue; }		// vertex is already colored
	
			bool neighborIsCurrent = false;
			for (auto voh_it = mesh.voh_iter(*v_it); voh_it.is_valid(); ++voh_it) {
				PolyMesh::VertexHandle vh1 = mesh.to_vertex_handle(*voh_it);
				PolyMesh::VertexHandle vh2 = mesh.to_vertex_handle(mesh.next_halfedge_handle(*voh_it));
				if (colorScheme[vh1.idx()] == colorSchemeIt || colorScheme[vh2.idx()] == colorSchemeIt) {
					neighborIsCurrent = true;
					break;
				}
			}
			if (neighborIsCurrent) { continue; }			// a neighboring vertex is already in this color
	
			colorScheme[v_it->idx()] = colorSchemeIt;
		}
		++colorSchemeIt;
	}

	int n_colors = *(std::max_element(colorScheme.begin(), colorScheme.end())) + 1;

	std::vector<int> n_color_vecs(n_colors, 0);
	for (int i = 0; i < colorScheme.size(); ++i) {
		if (colorScheme[i] > -1)
			++n_color_vecs[colorScheme[i]];
	}

	*coloredVertexIDs = new int[n_free_vertices];

	colorOffset = std::vector<int>(n_colors + 1, 0);
	for (int i = 1; i < n_colors; ++i) {
		colorOffset[i] = colorOffset[i - 1] + n_color_vecs[i - 1];
	}
	colorOffset[n_colors] = n_free_vertices;		// mark the end of the colored-vertices vector

													// add vertex ids
	std::vector<int>colorCounter(n_colors, 0);
	int interior_counter = 0;
	for (int i = 0; i < colorScheme.size(); ++i) {
		if (colorScheme[i] < 0) { continue; }
		(*coloredVertexIDs)[colorOffset[colorScheme[i]] + colorCounter[colorScheme[i]]++] = interior_counter++;
	}
}

template<int type = 0>
void discreteMeshOptimization( PolyMesh& mesh, const float grid_scale = 0.5f, int n_iter = 100) {
	
	int n_free_vertices = 0;
	int oneRingVecLength = 0;
#pragma omp parallel for reduction(+:n_free_vertices,oneRingVecLength)
	for (int i = 0; i < mesh.n_vertices(); ++i) {
		PolyMesh::VertexHandle vh = mesh.vertex_handle(i);
		if (mesh.is_boundary(vh)) { continue; }
		++n_free_vertices;

		for( auto voh : mesh.voh_range( vh ) ) {
			oneRingVecLength += 2;
		}
		++oneRingVecLength;		// additional count s.th. last element is again the first element
	}

	// convert OpenMesh to a basic structure
	real_cu* vertexPos = new real_cu[2 * mesh.n_vertices()];
	Vertex* vertices = new Vertex[n_free_vertices];
	int* oneRingVec = new int[oneRingVecLength];

	real_cu* vertexPos_d;
	Vertex* vertices_d;
	int* oneRingVec_d;
	int* coloredVertexIDs_d;

	int* coloredVertexIDs;
	std::vector<int> colorOffset;


#pragma omp parallel sections num_threads(2)
	{
#pragma omp section
		{
			gpuErrchk(hipMalloc((void**)&vertexPos_d, 2 * mesh.n_vertices() * sizeof(real_cu)));
			gpuErrchk(hipMalloc((void**)&vertices_d, n_free_vertices * sizeof(Vertex)));
			gpuErrchk(hipMalloc((void**)&oneRingVec_d, oneRingVecLength * sizeof(int)));
			gpuErrchk(hipMalloc((void**)&coloredVertexIDs_d, n_free_vertices * sizeof(int)));

			createColoring(mesh, n_free_vertices, &coloredVertexIDs, colorOffset);

			gpuErrchk(hipMemcpyAsync(coloredVertexIDs_d, coloredVertexIDs, n_free_vertices * sizeof(int), hipMemcpyHostToDevice));
		}
#pragma omp section 
		{
			copyOpenMeshData(mesh, vertexPos, vertices, oneRingVec);
		}
	}

	gpuErrchk(hipMemcpyAsync(vertexPos_d, vertexPos, 2 * mesh.n_vertices() * sizeof(real_cu), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(vertices_d, vertices, n_free_vertices * sizeof(Vertex), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(oneRingVec_d, oneRingVec, oneRingVecLength * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk( hipDeviceSynchronize() );
	
	const int n_colors = (int)colorOffset.size() - 1;


	constexpr real_cu affineFactor = 1.f / (real_cu)(NQ - 1);

	for (int i = 0; i < n_iter; ++i) {
		for (int cid = 0; cid < n_colors; ++cid) {
			const int nBlocks = colorOffset[cid + 1] - colorOffset[cid];
			const int nThreads = NQ * NQ / 2;
			//std::cout << "i = " << i << "  |  color = " << cid << std::endl;
			optimizeHierarchical << <nBlocks, nThreads >> >(coloredVertexIDs_d, colorOffset[cid], vertices_d, vertexPos_d, oneRingVec_d, affineFactor, grid_scale);
			gpuErrchk( hipDeviceSynchronize() );
		}
	}
	
	gpuErrchk( hipDeviceSynchronize() );
	hipMemcpy(vertexPos, vertexPos_d, 2 * mesh.n_vertices() * sizeof(real_cu), hipMemcpyDeviceToHost);

	hipFree(vertexPos_d);
	hipFree(vertices_d);
	hipFree(oneRingVec_d);
	hipFree(coloredVertexIDs_d);

	delete[] vertices;
	delete[] oneRingVec;
	delete[] coloredVertexIDs;
	
	// write vertex positions back to mesh
	for (auto v_it = mesh.vertices_begin(); v_it != mesh.vertices_end(); ++v_it) {
		int id = v_it->idx();
		TriMesh::Point p = { vertexPos[2 * id], vertexPos[2 * id + 1], 0.f };
		mesh.set_point(*v_it, p);
	}

	delete[] vertexPos;
}


template void discreteMeshOptimization<0>( PolyMesh&, const float, int );
template void discreteMeshOptimization<1>( PolyMesh&, const float, int );